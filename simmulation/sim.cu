#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <>
#include <crt/math_functions.hpp>
#include "pressuredata.h"

__device__ Transducer *d_transducers;
// __device__ float fData;
// __device__ int iData;


__global__ void simKernel(hipPitchedPtr pitchPtr, Vec3D gridCenter, float gridSize, size_t numOfTransducer, PressureParameter prePara)
{

    size_t pitchZ = pitchPtr.pitch/sizeof(float);

    float* magData = (float*)pitchPtr.ptr;

    int xId = threadIdx.x;
    int yId = blockIdx.x;
    int zId = blockIdx.y;

    float xRootPos = -(blockDim.x/2.0f)*gridSize + gridCenter.x;
    float yRootPos = -(gridDim.x/2.0f)*gridSize + gridCenter.y;
    float zRootPos = -(gridDim.y/2.0f)*gridSize + gridCenter.z;

    int idx = zId*gridDim.x*pitchZ + yId*pitchZ + xId;

    Vec3D targetPt(
        (xRootPos + xId*gridSize),
        (yRootPos + yId*gridSize),
        (zRootPos + zId*gridSize)
        );
    float reAcc=0;
    float imAcc=0;

    // if(xId == 1 && yId == 1 && zId ==0) {
    //     iData = idx;
    //     fData = xRootPos;
    // }

    for(int t=0; t< numOfTransducer; t++){
        Vec3D delta = targetPt - d_transducers[t].pos();
        float r = delta.length();
        // float theta = acos(delta.y*transducers[t].normal / r);
        float dot = delta.x*d_transducers[t].normal().x + delta.y*d_transducers[t].normal().y + delta.z*d_transducers[t].normal().z;
        float lenSq1 = r;
        float lenSq2 = d_transducers[t].normal().length();
        float theta = acos(dot/(lenSq1 * lenSq2));
        float phi = d_transducers[t].getRelativeRMSPhase();

        float Df=2 * j1f(prePara.waveNumber * d_transducers[t].diameter()/2*sinf(theta)) /
                   (prePara.waveNumber*d_transducers[t].diameter()/2*sinf(theta));
        float re = prePara.Vpp * d_transducers[t].P0() * Df / r * cos(phi+prePara.waveNumber*r);
        float im = prePara.Vpp * d_transducers[t].P0() * Df / r * sin(phi+prePara.waveNumber*r);

        reAcc += re;
        imAcc += im;

    }

    float mag = sqrtf(reAcc*reAcc + imAcc*imAcc);

    magData[idx]=mag;
}

extern "C" void magCalc(float* magData, std::vector<Transducer *> *transducers, PointGridData *pointGridData, PressureParameter pressureParameter)
{

    hipExtent extent = make_hipExtent(pointGridData->resolutionX()*sizeof(float)
                                      , pointGridData->resolutionY()
                                      , pointGridData->resolutionZ());
    hipPitchedPtr d_magData;
    hipMalloc3D(&d_magData, extent);

    Transducer* temp;
    hipMalloc((void**)&temp, transducers->size()*sizeof(Transducer));

    for (int i = 0; i < transducers->size(); ++i) {
        hipError_t trets = hipMemcpy(&temp[i], transducers->at(i), sizeof(Transducer), hipMemcpyHostToDevice);
        if(trets!= hipSuccess)
            std::cout << "error t: " << i << "--"<< hipGetErrorString(trets) <<"\n";
    }

    hipError_t rets = hipMemcpyToSymbol(HIP_SYMBOL(d_transducers), &temp, sizeof(temp));
    if(rets!= hipSuccess)
        std::cout << "error: " << "--"<< hipGetErrorString(rets) <<"\n";

    dim3 blockSize(pointGridData->resolutionX(), 1, 1);
    dim3 numblocks(pointGridData->resolutionY(), pointGridData->resolutionZ(), 1);

    simKernel<<<numblocks, blockSize>>>(d_magData,
                                        Vec3D(pointGridData->gridCenter().x,pointGridData->gridCenter().y, pointGridData->gridCenter().z),
                                        pointGridData->gridSize(),
                                        transducers->size(),
                                        pressureParameter);

    hipDeviceSynchronize();

    hipMemcpy3DParms memcpyParms = {0};
    memcpyParms.srcPtr=d_magData;
    memcpyParms.dstPtr.ptr=magData;
    memcpyParms.dstPtr.pitch=pointGridData->resolutionX() *sizeof(float);
    memcpyParms.dstPtr.xsize=pointGridData->resolutionX();
    memcpyParms.dstPtr.ysize=pointGridData->resolutionY();
    memcpyParms.extent.width=pointGridData->resolutionX() *sizeof(float);
    memcpyParms.extent.height=pointGridData->resolutionY();
    memcpyParms.extent.depth=pointGridData->resolutionZ();
    memcpyParms.kind=hipMemcpyDeviceToHost;

    // int idata = -1;
    // float fdata = -1;
    // hipMemcpyFromSymbol(&idata, HIP_SYMBOL(iData), sizeof(int));
    // hipMemcpyFromSymbol(&fdata, HIP_SYMBOL(fData), sizeof(float));
    // std::cout << "idata: " << idata <<"\n";
    // std::cout << "fdata: " << fdata <<"\n";

    hipError_t ret=hipMemcpy3D(&memcpyParms);
    if(ret!= hipSuccess)
        std::cout << "error " << hipGetErrorString(ret) <<"\n";

    hipFree(d_magData.ptr);
    hipFree(d_transducers);
    return;
}
